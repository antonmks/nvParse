#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/count.h>
#include <ctime>
#include "nvparse.h"

#ifdef _WIN64
#define atoll(S) _atoi64(S)
#include <windows.h>
#else
#include <unistd.h>
#endif


int main() {

	thrust::device_vector<int> d(3);
	thrust::host_vector<int> h(3);
	
	d[0] = 1;
	d[1] = 2;
	d[2] = 3;
	
	thrust::copy(d.begin(), d.end(), h.begin());
	
	for(int i = 0; i < 3; i++)
	std::cout << h[i] << std::endl;
	
    /*FILE* f = fopen("lineitem.tbl", "r" );
    fseek(f, 0, SEEK_END);
    long fileSize = ftell(f);
    thrust::device_vector<char> dev(fileSize);
    fseek(f, 0, SEEK_SET);
    char* buff;
    hipHostAlloc((void**) &buff, fileSize,hipHostMallocDefault);
    fread(buff, fileSize, 1, f);
    fclose(f);
    thrust::copy(buff, buff+fileSize, dev.begin());
    hipHostFree(buff);

    auto cnt = thrust::count(dev.begin(), dev.end(), '\n');
    std::cout << "There are " << cnt << " total lines in a file" << std::endl;

    thrust::device_vector<int> dev_pos(cnt+1);
    dev_pos[0] = -1;

    thrust::copy_if(thrust::make_counting_iterator((unsigned int)0), thrust::make_counting_iterator((unsigned int)fileSize),
                    dev.begin(), dev_pos.begin()+1, is_break());

    thrust::device_vector<char> dev_res1(cnt*15);
    thrust::fill(dev_res1.begin(), dev_res1.end(), 0);
    thrust::device_vector<char> dev_res2(cnt*15);
    thrust::fill(dev_res2.begin(), dev_res2.end(), 0);
    thrust::device_vector<char> dev_res3(cnt*15);
    thrust::fill(dev_res3.begin(), dev_res3.end(), 0);
    thrust::device_vector<char> dev_res4(cnt*15);
    thrust::fill(dev_res4.begin(), dev_res4.end(), 0);
    thrust::device_vector<char> dev_res5(cnt*15);
    thrust::fill(dev_res5.begin(), dev_res5.end(), 0);
    thrust::device_vector<char> dev_res6(cnt*15);
    thrust::fill(dev_res6.begin(), dev_res6.end(), 0);
    thrust::device_vector<char> dev_res7(cnt*15);
    thrust::fill(dev_res7.begin(), dev_res7.end(), 0);
    thrust::device_vector<char> dev_res8(cnt*15);
    thrust::fill(dev_res8.begin(), dev_res8.end(), 0);
    thrust::device_vector<char> dev_res9(cnt);
    thrust::fill(dev_res9.begin(), dev_res9.end(), 0);
    thrust::device_vector<char> dev_res10(cnt);
    thrust::fill(dev_res10.begin(), dev_res10.end(), 0);
    thrust::device_vector<char> dev_res11(cnt*10);
    thrust::fill(dev_res11.begin(), dev_res11.end(), 0);

    thrust::device_vector<char*> dest(11);
    dest[0] = thrust::raw_pointer_cast(dev_res1.data());
    dest[1] = thrust::raw_pointer_cast(dev_res2.data());
    dest[2] = thrust::raw_pointer_cast(dev_res3.data());
    dest[3] = thrust::raw_pointer_cast(dev_res4.data());
    dest[4] = thrust::raw_pointer_cast(dev_res5.data());
    dest[5] = thrust::raw_pointer_cast(dev_res6.data());
    dest[6] = thrust::raw_pointer_cast(dev_res7.data());
    dest[7] = thrust::raw_pointer_cast(dev_res8.data());
    dest[8] = thrust::raw_pointer_cast(dev_res9.data());
    dest[9] = thrust::raw_pointer_cast(dev_res10.data());
    dest[10] = thrust::raw_pointer_cast(dev_res11.data());

    thrust::device_vector<unsigned int> ind(11); //fields positions
    ind[0] = 0;
    ind[1] = 1;
    ind[2] = 2;
    ind[3] = 3;
    ind[4] = 4;
    ind[5] = 5;
    ind[6] = 6;
    ind[7] = 7;
    ind[8] = 8;
    ind[9] = 9;
    ind[10] = 10;

    thrust::device_vector<unsigned int> dest_len(11); //fields max lengths
    dest_len[0] = 15;
    dest_len[1] = 15;
    dest_len[2] = 15;
    dest_len[3] = 15;
    dest_len[4] = 15;
    dest_len[5] = 15;
    dest_len[6] = 15;
    dest_len[7] = 15;
    dest_len[8] = 1;
    dest_len[9] = 1;
    dest_len[10] = 10;


    thrust::device_vector<unsigned int> ind_cnt(1); //fields count
    ind_cnt[0] = 10;

    thrust::device_vector<char> sep(1);
    sep[0] = '|';

    std::clock_t start1 = std::clock();
    thrust::counting_iterator<unsigned int> begin(0);
    parse_functor ff((const char*)thrust::raw_pointer_cast(dev.data()),(char**)thrust::raw_pointer_cast(dest.data()), thrust::raw_pointer_cast(ind.data()),
                     thrust::raw_pointer_cast(ind_cnt.data()), thrust::raw_pointer_cast(sep.data()), thrust::raw_pointer_cast(dev_pos.data()), thrust::raw_pointer_cast(dest_len.data()));
    thrust::for_each(begin, begin + cnt, ff); // now dev_pos vector contains the indexes of new line characters

    thrust::device_vector<long long int> d_int(cnt);
    thrust::device_vector<double> d_float(cnt);
    std::cout<< "time0 " <<  ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) << '\n';

    //check the text results in dev_res array :
    for(int i = 0; i < 100; i++)
        std::cout << dev_res9[i];
    std ::cout << std::endl;

    for(int i = 0; i < 100; i++)
        std::cout << dev_res10[i];
    std ::cout << std::endl;

    //binary integer results
    ind_cnt[0] = 15;
    gpu_atoll atoll_ff((const char*)thrust::raw_pointer_cast(dev_res3.data()),(long long int*)thrust::raw_pointer_cast(d_int.data()),
                       thrust::raw_pointer_cast(ind_cnt.data()));
    thrust::for_each(begin, begin + cnt, atoll_ff);

    for(int i = 0; i < 10; i++)
        std::cout << d_int[i] << std::endl;

    std::cout <<  std::endl;

    //binary float results
    gpu_atof atof_ff((const char*)thrust::raw_pointer_cast(dev_res6.data()),(double*)thrust::raw_pointer_cast(d_float.data()),
                     thrust::raw_pointer_cast(ind_cnt.data()));
    thrust::for_each(begin, begin + cnt, atof_ff);

    std::cout.precision(10);
    for(int i = 0; i < 10; i++)
        std::cout << d_int[i] << std::endl;
	*/	

    return 0;

}
